#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <cinttypes>
#include <cstdint>
#include <cstdio>
#include <cstring>

#include "Shared.h"


// The GPU code.
__global__ void MultiplyStuffKernel(Limb *dest, const Limb *__restrict__ src)
{
	// Start of code.

	// Offset the two pointers depending on which block this is.
	dest += blockIdx.x * BLOCK_LIMB_COUNT;
	src += blockIdx.x * BLOCK_LIMB_COUNT;

	#define DECLARE_OUTPUT_ROUND(num) Limb var_t##num = 0

	#define DECLARE_OUTPUT_ROUND_9(r1, r2, r3, r4, r5, r6, r7, r8, r9) \
		DECLARE_OUTPUT_ROUND(r1); DECLARE_OUTPUT_ROUND(r2); DECLARE_OUTPUT_ROUND(r3); \
		DECLARE_OUTPUT_ROUND(r4); DECLARE_OUTPUT_ROUND(r5); DECLARE_OUTPUT_ROUND(r6); \
		DECLARE_OUTPUT_ROUND(r7); DECLARE_OUTPUT_ROUND(r8); DECLARE_OUTPUT_ROUND(r9);

	DECLARE_OUTPUT_ROUND(00);
	DECLARE_OUTPUT_ROUND_9(01, 02, 03, 04, 05, 06, 07, 08, 09);
	DECLARE_OUTPUT_ROUND_9(10, 11, 12, 13, 14, 15, 16, 17, 18);
	DECLARE_OUTPUT_ROUND_9(19, 20, 21, 22, 23, 24, 25, 26, 27);
	DECLARE_OUTPUT_ROUND_9(28, 29, 30, 31, 32, 33, 34, 35, 36);
	DECLARE_OUTPUT_ROUND_9(37, 38, 39, 40, 41, 42, 43, 44, 45);
	DECLARE_OUTPUT_ROUND_9(46, 47, 48, 49, 50, 51, 52, 53, 54);
	DECLARE_OUTPUT_ROUND_9(55, 56, 57, 58, 59, 60, 61, 62, 63);
	DECLARE_OUTPUT_ROUND(64);
	DECLARE_OUTPUT_ROUND(65);

	Limb var_multiplicand = 0;
	Limb var_m = 0;
	Limb var_carry = 0;
	Limb var_temp = 0;


	// Reduce subroutine.  Multiplies by R^-1 mod n.
	auto reduce = [&]()
	{
		// Reduce rounds.
		asm(
			// Get multiplicand used for reduction.
			"mul.lo.u32 %0, %1, " MODULUS_INVERSE_LOW ";\n\t"
			// The first round of reduction discards the result.
			"mad.lo.cc.u32 %2, %0, " MODULUS_WORD_00 ", %1;\n\t"
			"madc.hi.u32 %2, %0, " MODULUS_WORD_00 ", 0;\n\t" // note no % on 0
			:	"+r"(var_m), "+r"(var_t00), "+r"(var_carry));

		#define REDUCE_ROUND(prev, curr) \
			asm( \
				"mad.lo.cc.u32 %0, %2, " MODULUS_WORD_##curr ", %3;\n\t" \
				"madc.hi.u32 %3, %2, " MODULUS_WORD_##curr ", 0;\n\t" /* note no % on 0 */ \
				"add.cc.u32 %0, %0, %1;\n\t" \
				"addc.u32 %3, %3, 0;\n\t" /* note no % on 0*/ \
				:	"+r"(var_t##prev), "+r"(var_t##curr), "+r"(var_m), "+r"(var_carry))

		#define REDUCE_ROUND_9(r0, r1, r2, r3, r4, r5, r6, r7, r8, r9) \
			REDUCE_ROUND(r0, r1); REDUCE_ROUND(r1, r2); REDUCE_ROUND(r2, r3); \
			REDUCE_ROUND(r3, r4); REDUCE_ROUND(r4, r5); REDUCE_ROUND(r5, r6); \
			REDUCE_ROUND(r6, r7); REDUCE_ROUND(r7, r8); REDUCE_ROUND(r8, r9)

		REDUCE_ROUND_9(00, 01, 02, 03, 04, 05, 06, 07, 08, 09);
		REDUCE_ROUND_9(09, 10, 11, 12, 13, 14, 15, 16, 17, 18);
		REDUCE_ROUND_9(18, 19, 20, 21, 22, 23, 24, 25, 26, 27);
		REDUCE_ROUND_9(27, 28, 29, 30, 31, 32, 33, 34, 35, 36);
		REDUCE_ROUND_9(36, 37, 38, 39, 40, 41, 42, 43, 44, 45);
		REDUCE_ROUND_9(45, 46, 47, 48, 49, 50, 51, 52, 53, 54);
		REDUCE_ROUND_9(54, 55, 56, 57, 58, 59, 60, 61, 62, 63);

		asm(
			"add.cc.u32 %0, %1, %3;\n\t"
			"addc.u32 %1, %2, 0;\n\t" // note no % on 0
			:	"+r"(var_t63), "+r"(var_t64), "+r"(var_t65), "+r"(var_carry));
	};


	// Subroutine to write results.
	auto writeResults = [&](unsigned offset)
	{
		#define STORE_ROUND(num) dest[offset + ((1##num - 100) * NUM_THREADS)] = var_t##num;

		#define STORE_ROUND_9(r1, r2, r3, r4, r5, r6, r7, r8, r9) \
			STORE_ROUND(r1); STORE_ROUND(r2); STORE_ROUND(r3); \
			STORE_ROUND(r4); STORE_ROUND(r5); STORE_ROUND(r6); \
			STORE_ROUND(r7); STORE_ROUND(r8); STORE_ROUND(r9);

		STORE_ROUND(00);
		STORE_ROUND_9(01, 02, 03, 04, 05, 06, 07, 08, 09);
		STORE_ROUND_9(10, 11, 12, 13, 14, 15, 16, 17, 18);
		STORE_ROUND_9(19, 20, 21, 22, 23, 24, 25, 26, 27);
		STORE_ROUND_9(28, 29, 30, 31, 32, 33, 34, 35, 36);
		STORE_ROUND_9(37, 38, 39, 40, 41, 42, 43, 44, 45);
		STORE_ROUND_9(46, 47, 48, 49, 50, 51, 52, 53, 54);
		STORE_ROUND_9(55, 56, 57, 58, 59, 60, 61, 62, 63);
	};


	// First round moved out of main loop.
	// First round is special, because registers not set yet.
	// Multiplier for this round.
	var_multiplicand = src[(0 * NUM_THREADS) + threadIdx.x];

	asm(
		"mul.lo.u32 %0, %1, " MULTIPLIER_WORD_00 ";\n\t"
		"mul.hi.u32 %2, %1, " MULTIPLIER_WORD_00 ";\n\t"
		:	"+r"(var_t00), "+r"(var_multiplicand), "+r"(var_carry));

	#define MULTIPLY_FIRST_ROUND(num) \
		asm( \
			"mad.lo.cc.u32 %0, %1, " MULTIPLIER_WORD_##num ", %2;\n\t" \
			"madc.hi.u32 %2, %1, " MULTIPLIER_WORD_##num ", 0;\n\t" \
			:	"+r"(var_t##num), "+r"(var_multiplicand), "+r"(var_carry))

	#define MULTIPLY_FIRST_ROUND_9(r1, r2, r3, r4, r5, r6, r7, r8, r9) \
		MULTIPLY_FIRST_ROUND(r1); MULTIPLY_FIRST_ROUND(r2); MULTIPLY_FIRST_ROUND(r3); \
		MULTIPLY_FIRST_ROUND(r4); MULTIPLY_FIRST_ROUND(r5); MULTIPLY_FIRST_ROUND(r6); \
		MULTIPLY_FIRST_ROUND(r7); MULTIPLY_FIRST_ROUND(r8); MULTIPLY_FIRST_ROUND(r9);

	MULTIPLY_FIRST_ROUND_9(01, 02, 03, 04, 05, 06, 07, 08, 09);
	MULTIPLY_FIRST_ROUND_9(10, 11, 12, 13, 14, 15, 16, 17, 18);
	MULTIPLY_FIRST_ROUND_9(19, 20, 21, 22, 23, 24, 25, 26, 27);
	MULTIPLY_FIRST_ROUND_9(28, 29, 30, 31, 32, 33, 34, 35, 36);
	MULTIPLY_FIRST_ROUND_9(37, 38, 39, 40, 41, 42, 43, 44, 45);
	MULTIPLY_FIRST_ROUND_9(46, 47, 48, 49, 50, 51, 52, 53, 54);
	MULTIPLY_FIRST_ROUND_9(55, 56, 57, 58, 59, 60, 61, 62, 63);

	var_t64 = var_carry;
	var_t65 = 0;

	// Reduce - multiply by R^-1 mod n.
	reduce();


	// Main multiplication-reduction loop.
	for (unsigned i = 1; i < LIMB_COUNT; ++i)
	{
		// Multiplier for this round.
		var_multiplicand = src[(i * NUM_THREADS) + threadIdx.x];

		// Every round after the first is this way.
		asm(
			"mad.lo.cc.u32 %0, %1, " MULTIPLIER_WORD_00 ", %0;\n\t"
			"madc.hi.u32 %2, %1, " MULTIPLIER_WORD_00 ", 0;\n\t"  // note no % on this 0
			:	"+r"(var_t00), "+r"(var_multiplicand), "+r"(var_carry));

		#define MULTIPLY_ROUND(num) \
			asm( \
				"mad.lo.cc.u32 %0, %1, " MULTIPLIER_WORD_##num ", %0;\n\t" \
				"madc.hi.u32 %2, %1, " MULTIPLIER_WORD_##num ", 0;\n\t" /* note no % on 0 */ \
				"add.cc.u32 %0, %0, %3;\n\t" \
				"addc.u32 %3, %2, 0;\n\t" /* note no % on 0 */ \
				:	"+r"(var_t##num), "+r"(var_multiplicand), "+r"(var_temp), "+r"(var_carry))

		#define MULTIPLY_ROUND_9(r1, r2, r3, r4, r5, r6, r7, r8, r9) \
			MULTIPLY_ROUND(r1); MULTIPLY_ROUND(r2); MULTIPLY_ROUND(r3); \
			MULTIPLY_ROUND(r4); MULTIPLY_ROUND(r5); MULTIPLY_ROUND(r6); \
			MULTIPLY_ROUND(r7); MULTIPLY_ROUND(r8); MULTIPLY_ROUND(r9)

		MULTIPLY_ROUND_9(01, 02, 03, 04, 05, 06, 07, 08, 09);
		MULTIPLY_ROUND_9(10, 11, 12, 13, 14, 15, 16, 17, 18);
		MULTIPLY_ROUND_9(19, 20, 21, 22, 23, 24, 25, 26, 27);
		MULTIPLY_ROUND_9(28, 29, 30, 31, 32, 33, 34, 35, 36);
		MULTIPLY_ROUND_9(37, 38, 39, 40, 41, 42, 43, 44, 45);
		MULTIPLY_ROUND_9(46, 47, 48, 49, 50, 51, 52, 53, 54);
		MULTIPLY_ROUND_9(55, 56, 57, 58, 59, 60, 61, 62, 63);

		asm(
			"add.cc.u32 %0, %0, %1;\n\t"
			"addc.u32 %2, %2, 0;\n\t" // note no % on 0
			:	"+r"(var_t64), "+r"(var_carry), "+r"(var_t65));

		// Reduce - multiply by R^-1 mod n.
		reduce();
	}


	// Function to subtract the modulus if the result is greater than or equal
	// to the modulus.
	auto compareReduce1 = [&]()
	{
		// Compare against the modulus.
		asm(
			"sub.cc.u32 %0, %1, " MODULUS_WORD_00 ";\n\t"
			"addc.u32 %2, 0, 0;\n\t"  // extract carry flag to carry variable
			:	"+r"(var_temp), "+r"(var_t00), "+r"(var_carry));

		#define COMPARE_ROUND(regnum, wordnum) \
			"subc.cc.u32 %0, %" #regnum ", " MODULUS_WORD_##wordnum ";\n\t"

		#define COMPARE_ROUND_9(r1, r2, r3, r4, r5, r6, r7, r8, r9) \
			asm( \
				"add.cc.u32 %0, %10, 0xFFFFFFFFU;\n\t" /* put carry variable into carry flag */ \
				COMPARE_ROUND(1, r1) COMPARE_ROUND(2, r2) COMPARE_ROUND(3, r3) \
				COMPARE_ROUND(4, r4) COMPARE_ROUND(5, r5) COMPARE_ROUND(6, r6) \
				COMPARE_ROUND(7, r7) COMPARE_ROUND(8, r8) COMPARE_ROUND(9, r9) \
				"addc.u32 %10, 0, 0;\n\t"  /* extract carry flag to carry variable */ \
				:	"+r"(var_temp), \
					"+r"(var_t##r1), "+r"(var_t##r2), "+r"(var_t##r3), \
					"+r"(var_t##r4), "+r"(var_t##r5), "+r"(var_t##r6), \
					"+r"(var_t##r7), "+r"(var_t##r8), "+r"(var_t##r9), \
					"+r"(var_carry))

		COMPARE_ROUND_9(01, 02, 03, 04, 05, 06, 07, 08, 09);
		COMPARE_ROUND_9(10, 11, 12, 13, 14, 15, 16, 17, 18);
		COMPARE_ROUND_9(19, 20, 21, 22, 23, 24, 25, 26, 27);
		COMPARE_ROUND_9(28, 29, 30, 31, 32, 33, 34, 35, 36);
		COMPARE_ROUND_9(37, 38, 39, 40, 41, 42, 43, 44, 45);
		COMPARE_ROUND_9(46, 47, 48, 49, 50, 51, 52, 53, 54);
		COMPARE_ROUND_9(55, 56, 57, 58, 59, 60, 61, 62, 63);

		// If necessary, subtract.
		// carry is 1 if above or equal, 0 if not (6502 semantics, not x86).
		if ((var_carry != 0) || (var_t64 != 0))
		{
			asm(
				"sub.cc.u32 %1, %1, " MODULUS_WORD_00 ";\n\t"
				"addc.u32 %2, 0, 0;\n\t"  // extract carry flag to carry variable
				:	"+r"(var_temp), "+r"(var_t00), "+r"(var_carry));

			#define SUBTRACT_ROUND(regnum, wordnum) \
				"subc.cc.u32 %" #regnum ", %" #regnum ", " MODULUS_WORD_##wordnum ";\n\t"

			#define SUBTRACT_ROUND_9(r1, r2, r3, r4, r5, r6, r7, r8, r9) \
				asm( \
					"add.cc.u32 %0, %10, 0xFFFFFFFFU;\n\t" /* put carry variable into carry flag */ \
					SUBTRACT_ROUND(1, r1) SUBTRACT_ROUND(2, r2) SUBTRACT_ROUND(3, r3) \
					SUBTRACT_ROUND(4, r4) SUBTRACT_ROUND(5, r5) SUBTRACT_ROUND(6, r6) \
					SUBTRACT_ROUND(7, r7) SUBTRACT_ROUND(8, r8) SUBTRACT_ROUND(9, r9) \
					"addc.u32 %10, 0, 0;\n\t"  /* extract carry flag to carry variable */ \
					:	"+r"(var_temp), \
						"+r"(var_t##r1), "+r"(var_t##r2), "+r"(var_t##r3), \
						"+r"(var_t##r4), "+r"(var_t##r5), "+r"(var_t##r6), \
						"+r"(var_t##r7), "+r"(var_t##r8), "+r"(var_t##r9), \
						"+r"(var_carry))

			SUBTRACT_ROUND_9(01, 02, 03, 04, 05, 06, 07, 08, 09);
			SUBTRACT_ROUND_9(10, 11, 12, 13, 14, 15, 16, 17, 18);
			SUBTRACT_ROUND_9(19, 20, 21, 22, 23, 24, 25, 26, 27);
			SUBTRACT_ROUND_9(28, 29, 30, 31, 32, 33, 34, 35, 36);
			SUBTRACT_ROUND_9(37, 38, 39, 40, 41, 42, 43, 44, 45);
			SUBTRACT_ROUND_9(46, 47, 48, 49, 50, 51, 52, 53, 54);
			SUBTRACT_ROUND_9(55, 56, 57, 58, 59, 60, 61, 62, 63);
		}
	};


	// Do one iteration of the subtraction.
	compareReduce1();

	// Write results, which is a*b*R mod n.
	writeResults((0 * TOTAL_LIMB_COUNT) + threadIdx.x);

	// Multiply by R^-1 again so that the actual answer is visible.
	var_t64 = 0;
	var_t65 = 0;
	for (unsigned i = 0; i < LIMB_COUNT; ++i)
	{
		reduce();
	}
	compareReduce1();

	// Write the non-Montgomery results.
	writeResults((1 * TOTAL_LIMB_COUNT) + threadIdx.x);

#undef DECLARE_OUTPUT_ROUND
#undef DECLARE_OUTPUT_ROUND_9
#undef MULTIPLY_FIRST_ROUND
#undef MULTIPLY_FIRST_ROUND_9
#undef MULTIPLY_ROUND
#undef MULTIPLY_ROUND_9
#undef REDUCE_ROUND
#undef REDUCE_ROUND_9
#undef COMPARE_ROUND
#undef COMPARE_ROUND_9
#undef SUBTRACT_ROUND
#undef SUBTRACT_ROUND_9
#undef STORE_ROUND
#undef STORE_ROUND_9
}


// Constructor to just null out the pointers.
GPUState::GPUState()
{
	d_buffers[0] = nullptr;
	d_buffers[1] = nullptr;
}


// Destructor to free resources.
GPUState::~GPUState()
{
	if (d_buffers[0])
	{
		hipFree(d_buffers[0]);
		d_buffers[0] = nullptr;
	}

	if (d_buffers[1])
	{
		hipFree(d_buffers[1]);
		d_buffers[1] = nullptr;
	}

	hipDeviceReset();
}


// Main initialization function.
hipError_t GPUState::Initialize()
{
	hipError_t status;

	// Choose which GPU to run on, change this on a multi-GPU system.
	status = hipSetDevice(0);
	if (status != hipSuccess)
	{
		return status;
	}

	// Allocate the two buffers.
	status = hipMalloc(&d_buffers[0], sizeof(Limb[TOTAL_LIMB_COUNT * 2]));
	if (status != hipSuccess)
	{
		d_buffers[0] = nullptr;
		return status;
	}

	status = hipMalloc(&d_buffers[1], sizeof(Limb[TOTAL_LIMB_COUNT * 2]));
	if (status != hipSuccess)
	{
		d_buffers[1] = nullptr;
		return status;
	}

	return hipSuccess;
}


// Reseeds the state for a new round.
hipError_t GPUState::Reseed(unsigned currentSrc, const Limb seed[TOTAL_LIMB_COUNT])
{
	// Copy to the specified buffer.
	return hipMemcpy(d_buffers[currentSrc], seed, sizeof(Limb[TOTAL_LIMB_COUNT]), hipMemcpyHostToDevice);
}


// Execute the math operation.
hipError_t GPUState::Execute(unsigned currentSrc, Limb output[TOTAL_LIMB_COUNT])
{
	hipError_t status;

	// Debug information.
	hipFuncAttributes attributes;
	status = hipFuncGetAttributes(&attributes, reinterpret_cast<const void*>(reinterpret_cast<void *>(MultiplyStuffKernel)));
	if (status != hipSuccess)
	{
		return status;
	}

	static bool s_printed = true; //false;
	if (!s_printed)
	{
	#define OUTPUT_DEBUG(field, formatter) std::printf(#field " = " formatter "\n", field)
		OUTPUT_DEBUG(attributes.sharedSizeBytes, "%zu");
		OUTPUT_DEBUG(attributes.constSizeBytes, "%zu");
		OUTPUT_DEBUG(attributes.localSizeBytes, "%zu");
		OUTPUT_DEBUG(attributes.maxThreadsPerBlock, "%d");
		OUTPUT_DEBUG(attributes.numRegs, "%d");
		OUTPUT_DEBUG(attributes.ptxVersion, "%d");
		OUTPUT_DEBUG(attributes.binaryVersion, "%d");
		OUTPUT_DEBUG(attributes.cacheModeCA, "%d");
		s_printed = true;
	}

	// Execute operation.
	MultiplyStuffKernel<<<NUM_BLOCKS, NUM_THREADS>>>(
		static_cast<Limb *>(d_buffers[currentSrc ^ 1]),
		static_cast<Limb *>(d_buffers[currentSrc]));

	// Check for any errors launching the kernel
	status = hipGetLastError();
	if (status != hipSuccess)
	{
		return status;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	status = hipDeviceSynchronize();
	if (status != hipSuccess)
	{
		return status;
	}

	// Copy result to host.
	return hipMemcpy(output, static_cast<char *>(d_buffers[currentSrc ^ 1]) + sizeof(Limb[TOTAL_LIMB_COUNT]), 
		sizeof(Limb[TOTAL_LIMB_COUNT]), hipMemcpyDeviceToHost);
}
